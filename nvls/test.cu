#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <stdio.h>
// #include <nccl.h>
#include <unistd.h>
#include <cudaTypedefs.h>
#include "ipcsocket.cc"

#define CUCHECK(cmd) do {               \
    auto err = cmd;                     \
    if( err != 0 ) {                    \
        printf("Cuda failure %d: Line %d", err, __LINE__); \
    }                                   \
} while(false)

//AR kernel snippet for sm_90 only

#define MULTIMEM_ST(val, ptr)                                                  \
  asm volatile("multimem.st.global.v4.f32 [%0], {%1,%2,%3,%4};" ::"l"(ptr),    \
               "r"(val.x), "r"(val.y), "r"(val.z), "r"(val.w)                  \
               : "memory");
//specific PTX for fp16 reduction. bf16 would be multimem.ld_reduce.global.add.v4.bf16x2 etc
#define MULTIMEM_LD(val, ptr)                                                  \
  asm("multimem.ld_reduce.global.add.v4.f32 {%0,%1,%2,%3}, [%4];"            \
      : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)                     \
      : "l"(ptr)                                                               \
      : "memory");

__global__ void testing2(float* uc_ptr){
  uc_ptr[0] = 1.0;
  printf("ptr -> %f\n", uc_ptr[0]);
}


__global__ void testing(float* mc_ptr, int numlines, int myrank, int RANKS){
  //for allreduce we dont even need an UC pointer. just using same mc_ptr for in-place reduction
  //line is assumed to be 16B 4 ints of 8 halves
  const int start_elem =  threadIdx.x + blockDim.x * (myrank + RANKS * blockIdx.x);
  const int end_elem = max(start_elem, numlines);
  const int lineoffset = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  const int loop_step0 = (blockDim.x * gridDim.x) * 4;
  __syncthreads();
  printf("start %d, end %d step %d\n", start_elem, end_elem, loop_step0);
    for (int line = start_elem; line < end_elem; line += loop_step0) {
      uint4 val;
      MULTIMEM_LD(val, mc_ptr + (lineoffset + line));
      printf("val %f\n", *(float*)&(val.x));
      MULTIMEM_ST(val, mc_ptr + (lineoffset + line));
    }
  __syncthreads();
}

int main(){
  int myrank, nranks;
  MPI_Init(NULL, NULL);
  MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
  MPI_Comm_size(MPI_COMM_WORLD, &nranks);
 
  hipSetDevice(myrank);
  hipError_t res;

  size_t size = 1024*1024*512*3;
  hipMemAllocationHandleType handleType = hipMemHandleTypePosixFileDescriptor;
 
  CUmulticastObjectProp mcProp = {};
  mcProp.numDevices = nranks;
  mcProp.size = size;
  mcProp.handleTypes = handleType;
 
  size_t minGran, gran;
  gran = 0;
  minGran = 0;
  CUCHECK(cuMulticastGetGranularity(&minGran, &mcProp, CU_MULTICAST_GRANULARITY_MINIMUM));
  CUCHECK(cuMulticastGetGranularity(&gran, &mcProp, CU_MULTICAST_GRANULARITY_RECOMMENDED));

  // printf("gran = %lld, minGrad = %lld\n", gran, minGran); 
  size_t mcSize = ((size+gran-1)/gran)*gran;
  mcProp.size = mcSize;

  hipMemGenericAllocationHandle_t handle;
  //only one rank creates the multicast object
  if(!myrank) CUCHECK(cuMulticastCreate(&handle, &mcProp));
 
  int fd, peerfd;
  fd = 0;
  peerfd = 0;
  if(!myrank) CUCHECK(hipMemExportToShareableHandle(&fd, handle, handleType, 0 /*flags*/));
 
  //some ugly UDS business
  // Borrow ipcsocket.{c,h} from nccl code
  //in cuda 12.4 new fabric handle type is available so instead it would be possible to use MPI_Allgather for the exported handles
  // moreover it would the only way to do it on GraceHopper systems, since UDS is limited to single Unix node
 
  volatile uint32_t abortFlag = 0;
 struct ncclIpcSocket ipcSock = { 0 };
 uint64_t opId=0xdeadcafebeef;
  // ncclResult_t ret = ncclSuccess;

  ncclIpcSocketInit(&ipcSock, myrank, (uint64_t)opId, &abortFlag);
  MPI_Barrier(MPI_COMM_WORLD);
  if(!myrank) {
    for(int p=1;p<nranks;p++) {
      ncclIpcSocketSendFd(&ipcSock, fd, p, (uint64_t)opId);
    } 
  } else {
      ncclIpcSocketRecvFd(&ipcSock, &peerfd);
  }
  ncclIpcSocketClose(&ipcSock);

  printf("fd = %d peerfd = %d\n", fd, peerfd);
  // MPI_Bcast(&fd, sizeof(fd), MPI_CHAR, 0, MPI_COMM_WORLD);
  //everyone else would now have same multicast object
  if(myrank)  CUCHECK(hipMemImportFromShareableHandle(&handle, (void *)peerfd, handleType));
 
//  if(myrank)
//    close(peerfd);
//  else
    close(fd);
  //end of ugly UDS business
#if 1
  //everyone adds device(s), no syncs required, just need to ensure bindmem happens after all this is called
  int mydev = myrank;
  CUCHECK(cuMulticastAddDevice(handle, mydev));
  MPI_Barrier(MPI_COMM_WORLD);
 
  hipMemGenericAllocationHandle_t memhandle;
  hipMemAllocationProp prop = {};
  prop.type = hipMemAllocationTypePinned;
  prop.location.type = hipMemLocationTypeDevice;
  prop.location.id = mydev;
  prop.requestedHandleTypes = handleType;
 
  //allocate physical memory (data buffer)
  CUCHECK(hipMemCreate(&memhandle, size, &prop, 0 /*flags*/));
 
  //everyone binds memory to the multicast
  CUCHECK(cuMulticastBindMem(handle, 0 /*mcOffset*/, memhandle, 0 /*memOffset*/, size, 0));
  MPI_Barrier(MPI_COMM_WORLD);
  //usual VA business: map both MC and PA to two different VA addresses
  void* uc_va;
  void* mc_va;
    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = mydev;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
 
    // Map a VA to UC space
    CUCHECK(hipMemAddressReserve((hipDeviceptr_t*)&uc_va, size, minGran, 0U, 0));
    hipMemset(uc_va, 0, size);
    CUCHECK(hipMemMap((hipDeviceptr_t)uc_va, size, 0, memhandle, 0));
    // set access on UC address
    CUCHECK(hipMemSetAccess((hipDeviceptr_t)uc_va, size, &accessDesc, 1));
 
  // Map a VA to MC space
  CUCHECK(hipMemAddressReserve((hipDeviceptr_t*)&mc_va, mcSize, minGran, 0U, 0));
  CUCHECK(hipMemMap((hipDeviceptr_t)mc_va, mcSize, 0, handle, 0));
  // set access on MC address
  CUCHECK(hipMemSetAccess((hipDeviceptr_t)mc_va, mcSize, &accessDesc, 1));
#endif
  printf("Yoooo\n");
  testing2<<<1, 1>>>((float*)mc_va);
  hipDeviceSynchronize();
  MPI_Barrier(MPI_COMM_WORLD);
  testing<<<1, 1>>>((float*)mc_va, 1, myrank, nranks);
  hipDeviceSynchronize();
  MPI_Barrier(MPI_COMM_WORLD);
  MPI_Finalize();
} 
//........
 

